#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>

#include<windows.h>
//for linux
//#include<unistd.h>


struct Runner {
	int dist;
	int vel;
};

struct Team {
	Runner* runners;
	int curRunner;
	int id;
};


hipError_t createTeamsWithCuda(Team* teams, Runner* runners, const int size);

hipError_t simulateRaceWithCuda(Team* teams, Runner* runners, int* finished_team_count, int* placements, const int size);

__device__ int lock = 0;

__global__ void createTeamKernel(Team* teams, Runner* runners)
{
	int i = threadIdx.x;
	//RUN_LIMIT
	int size = 4;
	teams[i].runners = &runners[i * size];
	for (int j = 0; j < size; j++)
	{
		teams[i].runners[j].dist = j * 100;
		//At first everybody is at stop.
		teams[i].runners[j].vel = 0;
	}
	teams[i].id = i + 1;
	teams[i].curRunner = 0;

}



__global__ void simulateRaceKernel(Team* teams, Runner* runners, int* finished_team_count, int* placements, int rand_seed)
{
	int i = threadIdx.x;
	teams[i].runners = &runners[i * 4];
	int* curRunner = &teams[i].curRunner;

	if (*curRunner == 4)
	{
		//This team has ended the race.
		return;
	}

	if (teams[i].runners[*curRunner].dist < (*curRunner + 1) * 100)
	{
		teams[i].runners[*curRunner].dist += teams[i].runners[*curRunner].vel;
	}

	if (teams[i].runners[*curRunner].dist >= (*curRunner + 1) * 100)
	{
		teams[i].runners[*curRunner].vel = 0;
		*curRunner += 1;
	}

	if (*curRunner == 4)
	{

		//Race condition can not be paralel
		//Because threads can get asynchronous counts
		//Other solution in my mind was before calling kernel for every 400 threads
		//Count the curRunner == 4 flag to check if all 400 teams have finished the race
		//But that meant every second (Each Run) program would need to count from scratch
		//So This SingleThreaded madness must take place here.
		//Because Some teams finish very early and at most there was only 30-40 threads on lock.

		bool leaveLoop = false;
		while (!leaveLoop) {
			if (atomicExch(&lock, 1u) == 0u) {
				placements[*finished_team_count] = i;
				if (*finished_team_count == 0)
				{
					printf("\nFirst team to arrive finish line is Team %d\n", i + 1);
					printf("-----------------------------------------------\n");
					for (int j = 0; j < 4; j++)
					{
						printf("Team %d Runner %d VEL:%d DIST:%d\n", i + 1, j + 1, teams[i].runners[j].vel, teams[i].runners[j].dist);
					}
					printf("-----------------------------------------------\n");
				}
				*finished_team_count += 1;
				leaveLoop = true;
				atomicExch(&lock, 0u);
			}
		}

	}


	else
	{
		hiprandState_t state;
		hiprand_init(rand_seed, i, 0, &state);

		teams[i].runners[*curRunner].vel = hiprand_uniform(&state) * 5 + 1;
	}
}





int main(int argc, char* argv[])
{
	//400 olucak ama test için 5
	const int TEAM_SIZE = 400;

	//pointer for objects
	Team* teams = new Team[TEAM_SIZE];
	Runner* runners = new Runner[TEAM_SIZE * 4];
	//Pointer for placements
	int* placements = new int[TEAM_SIZE];


	hipError_t cudaStatus = createTeamsWithCuda(teams, runners, TEAM_SIZE);

	//Pointers must be reassigned because 
	//Pointer values on the objects are for gpu memory (video-ram)
	//They are needed to be repointed to cpu memory (ram or virtual ram)
	for (int i = 0; i < TEAM_SIZE; i++) {
		teams[i].runners = &runners[i * 4];
	}


	int* consoleTeams;
	int consoleSize = 0;
	int finished_team_count = 0;

	if (argc <= 1)
	{
		consoleTeams = new int[TEAM_SIZE];
		printf("No arguments are passed while running the program.\nPlease state which teams will be shown on the console.\n");
		printf("All numbers must be seperated by space\n");
		do {
			scanf("%d", &consoleTeams[consoleSize++]);
		} while (getchar() != '\n' && consoleSize < TEAM_SIZE);
	}
	else
	{
		consoleTeams = new int[argc - 1];
		for (int i = 1; i < argc; i++)
		{
			sscanf(argv[i], "%d", &consoleTeams[consoleSize++]);
		}
	}
	for (int i = 0; i < consoleSize; i++)
	{
		if (consoleTeams[i] <= 0)
		{
			printf("Can't give an argument below or equal to 0 or NaN. Teams start at 1.");
			exit(-1);
		}
		else if (consoleTeams[i] > TEAM_SIZE)
		{
			printf("Can't select non existent team");
			exit(-2);
		}

	}

	while (finished_team_count < TEAM_SIZE)
	{
		simulateRaceWithCuda(teams, runners, &finished_team_count, placements, TEAM_SIZE);
		//Pointers must be reassigned because 
		//Pointer values on the objects are for gpu memory (video-ram)
		//They are needed to be repointed to cpu memory (ram or virtual ram)
		for (int i = 0; i < TEAM_SIZE; i++)
		{
			teams[i].runners = &runners[i * 4];
		}

		for (int i = 0; i < consoleSize; i++)
		{
			printf("-------------------------\n");
			int outTeam = consoleTeams[i];
			for (int j = 0; j < 4; j++)
			{
				printf("Team %d Runner %d VEL:%d DIST:%d\n", outTeam, j + 1, teams[outTeam - 1].runners[j].vel, teams[outTeam - 1].runners[j].dist);
			}
		}


		printf("-------------------------\n");
		printf("Finished Team Count:%d\n", finished_team_count);
		printf("|||||||||||||||||||||||||\n");

		//Sleep function in windows is in milliseconds
		//1000 olucak
		Sleep(1 * 1000);
		//For linux based
		// It is in seconds for linux.
		//sleep(1);
	}

	printf("Race has ended The Results are\n");;

	for (int i = 0; i < TEAM_SIZE; i++)
	{
		printf("%d PLACE: TEAM %d\n", i + 1, teams[placements[i]].id);
	}



	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	free(teams);
	free(runners);
	free(placements);
	free(consoleTeams);
	return 0;
}

hipError_t simulateRaceWithCuda(Team* teams, Runner* runners, int* finished_team_count, int* placements, const int size)
{
	Team* dev_teams;
	Runner* dev_runners;
	int* dev_placements;
	int* dev_count;
	hipError_t cudaStatus;

	srand(time(NULL));
	int rand_seed = rand() % 500 + 1000;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_placements, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_placements, placements, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}



	cudaStatus = hipMalloc((void**)&dev_teams, size * sizeof(Team));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_teams, teams, size * sizeof(Team), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_runners, size * sizeof(Runner) * 4);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_runners, runners, size * sizeof(Runner) * 4, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_count, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_count, finished_team_count, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on flag failed!\n");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	simulateRaceKernel << <1, size >> > (dev_teams, dev_runners, dev_count, dev_placements, rand_seed);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "simulateRaceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching simulateRaceKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.

	cudaStatus = hipMemcpy(teams, dev_teams, size * sizeof(Team), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on Teams failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(runners, dev_runners, size * sizeof(Runner) * 4, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on Runners failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(finished_team_count, dev_count, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on count failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(placements, dev_placements, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on placements failed!");
		goto Error;
	}


Error:
	hipFree(dev_teams);
	hipFree(dev_runners);
	hipFree(dev_count);
	hipFree(dev_placements);

	return cudaStatus;

}


hipError_t createTeamsWithCuda(Team* teams, Runner* runners, const int size)
{
	Team* dev_teams;
	Runner* dev_runners;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_teams, size * sizeof(Team));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_teams, teams, size * sizeof(Team), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_runners, size * sizeof(Runner) * 4);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_runners, runners, size * sizeof(Runner) * 4, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	// Launch a kernel on the GPU with one thread for each element.
	createTeamKernel << <1, size >> > (dev_teams, dev_runners);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "createTeamKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//printf("%d", dev_teams[0].runners[0].dist);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching createTeamKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.

	cudaStatus = hipMemcpy(teams, dev_teams, size * sizeof(Team), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on Teams failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(runners, dev_runners, size * sizeof(Runner) * 4, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on Runners failed!");
		goto Error;
	}




Error:
	hipFree(dev_teams);
	hipFree(dev_runners);

	return cudaStatus;

}
